#include "hip/hip_runtime.h"
/*
* This is a program to test 
* matrix multiplication efficiency
* On CPU and GPU
*
* @Author: zutterhao Nanjing University
* @Date: 2019/5/24
*/
#include <iostream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#include <stdlib.h>
#include <Eigen/Core>
#include <Eigen/Cholesky>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
using namespace std;

const unsigned int THREAD_PER_BLOCK = 1024;
const int BLOCK_SIZE = 32;



/*
* @func: print matrix(small size)
*
* @para: matrix print matrix
*/
__host__ void printMatrix(float* matrix,unsigned int row,unsigned int col)
{
	if (col > 16 || row > 16)
	{
		cout << "Matrix size to large !" << endl;
		return;
	}
	cout << endl << "*******************************************" << endl
		<< "Print Matrix :" << endl;
	for (size_t i = 0; i < row; ++i)
	{
		for (size_t j = 0; j < col; ++j)
		{
			cout << matrix[i * col + j] << " ";
		}
		cout << endl;
	}
	cout << "*******************************************" << endl;
	return;
}


/*
* @func: matrix multiplication on cpu
*
* @para: m_a  left multi matrix
*		 m_b  right multi matrix
*		 m_r  result matrix
*		 m    left matrix rows
*        n    left matrix cols
*        k    right matrix cols
*/
__host__ void matrixMulOnCPU(float* m_a, float* m_b, float* m_r, unsigned int m, unsigned int n, unsigned int k)
{
	if (m <= 0 || n <= 0 || k <= 0)
	{
		cout << "Matrix size error !" << endl;
		return;
	}
	clock_t multi_start = clock();
	for (size_t i = 0; i < m; ++i)
	{
		for (size_t j = 0; j < k; ++j)
		{
			float tmp = 0.0f;
			unsigned int index = i * k + j;
			for (size_t t = 0; t < n; ++t)
			{
				tmp += m_a[n * i + t] * m_b[k * t + j];
			}
			m_r[index] = tmp;
		}
	}
	clock_t multi_end = clock();
	cout << endl << "*******************************************" << endl
		<< "Matrix A size : " << m << " * " << n << endl
		<< "Matrix B size : " << n << " * " << k << endl
		<< "CPUMultiTime : " << (multi_end - multi_start) << " ms" << endl
		<< "*******************************************"
		<< endl;
	return;
}


/*
* @func: generate random matrix
*
* @para: m_a  left multi matrix
*		 m_b  right multi matrix
*		 m    left matrix rows
*        n    left matrix cols
*        k    right matrix cols
*/
__host__ void genMatrixValue(float** m_a, float** m_b, float** m_r, unsigned int m, unsigned int n, unsigned int k)
{
	if (m <= 0 || n <= 0 || k <= 0)
	{
		cout << "Generate matrix size error !" << endl;
		return;
	}
	clock_t gen_start = clock();
	// malloc memory
	size_t m_a_size = m * n * sizeof(float);
	size_t m_b_size = n * k * sizeof(float);
	size_t m_r_size = m * k * sizeof(float);

	*m_a = (float *)malloc(m_a_size);
	*m_b = (float *)malloc(m_b_size);
	*m_r = (float *)malloc(m_r_size);

	// generate m_a
	for (size_t i = 0; i < m; ++i)
	{
		for (size_t j = 0; j < n; ++j)
		{
			(*m_a)[n * i + j] = float(n * i + j);
		}
	}
	// generate m_b
	for (size_t i = 0; i < n; ++i)
	{
		for (size_t j = 0; j < k; ++j)
		{
			(*m_b)[k * i + j] = float(n * k - k * i -j - 1);
		}
	}
	clock_t gen_end = clock();
	cout << "Generate matrix_a and matrix_b succeed ! Time : " << (gen_end - gen_start) << " ms" << endl;
}

/*
* @func: matrix multiplication use eigen 
*
* @para: m_a  left multi matrix
*		 m_b  right multi matrix
*		 m_r  result matrix
*		 m    left matrix rows
*        n    left matrix cols
*        k    right matrix cols
*/
__host__ void matrixMulUseEigen(float* m_a, float* m_b, float* m_r, unsigned int m, unsigned int n, unsigned int k)
{
	if (m <= 0 || n <= 0 || k <= 0)
	{
		cout << "Matrix size error !" << endl;
		return;
	}
	// Matrix variable
	Eigen::MatrixXd left = Eigen::MatrixXd::Zero(m, n);
	Eigen::MatrixXd right = Eigen::MatrixXd::Zero(n, k);
	Eigen::MatrixXd result;

	// copy matrix from array to eigen matrix
	for (size_t i = 0; i < m; ++i)
	{
		for (size_t j = 0; j < n; ++j)
		{
			left(i, j) = m_a[i * n + j];
		}
	}
	for (size_t i = 0; i < n; ++i)
	{
		for (size_t j = 0; j < k; ++j)
		{
			right(i, j) = m_b[i * k + j];
		}
	}
	clock_t eigen_start = clock();
	result = left * right;
	clock_t eigen_end = clock();
	
	for (size_t i = 0; i < m; ++i)
	{
		for (size_t j = 0; j < k; ++j)
		{
			m_r[i * k + j] = result(i, j);
		}
	}
	cout << endl << "*******************************************" << endl
		<< "Matrix A size : " << m << " * " << n << endl
		<< "Matrix B size : " << n << " * " << k << endl
		<< "EigenMultiTime : " << (eigen_end - eigen_start) << " ms" << endl
		<< "*******************************************"
		<< endl;
	return;
}

/*
* @func: matrix multiplication on gpu
*
* @para: m_a  left multi matrix
*		 m_b  right multi matrix
*		 m_r  result matrix
*		 m    left matrix rows
*        n    left matrix cols
*        k    right matrix cols
*/
__global__ void matrixMulOnGPU(float* m_a, float* m_b, float* m_r, unsigned int m, unsigned int n, unsigned int k)
{
	int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= m * k)
		return;

	int row = threadId / k;
	int col = threadId % k;

	m_r[threadId] = 0;
	for (size_t i = 0; i < n; ++i)
	{
		m_r[threadId] += m_a[row * n + i] * m_b[i * k + col];
	}
}
/*
* @func: matrix multiplication use gpu accelerate
*
* @para: m_a  left multi matrix
*		 m_b  right multi matrix
*		 m_r  result matrix
*		 m    left matrix rows
*        n    left matrix cols
*        k    right matrix cols
*/
__host__ void matrixMulUseGPU(float* m_a, float* m_b, float* m_r, unsigned int m, unsigned int n, unsigned int k)
{
	if (m <= 0 || n <= 0 || k <= 0)
	{
		cout << "Matrix size error !" << endl;
		return;
	}
	float *MA, *MB, *MR;

	unsigned int a_num = m * n;
	unsigned int b_num = n * k;
	unsigned int r_num = m * k;

	// Allocated memory
	hipMalloc((void**)&MA, a_num * sizeof(float));
	hipMalloc((void**)&MB, b_num * sizeof(float));
	hipMalloc((void**)&MR, r_num * sizeof(float));

	// copy data from cpu to gpu
	clock_t todevice_start = clock();
	hipMemcpy(MA, m_a, a_num * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(MB, m_b, b_num * sizeof(float), hipMemcpyHostToDevice);
	clock_t todevice_end = clock();

	// calculate grids and blocks
	unsigned int thread_num = min(r_num, THREAD_PER_BLOCK);
	unsigned int block_num = (r_num % thread_num != 0) ? (r_num / thread_num + 1) : (r_num / thread_num);
	
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, NULL);
	matrixMulOnGPU << <block_num, thread_num >> > (MA, MB, MR, m, n, k);
	hipDeviceSynchronize();
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	float gpu_time = 0.0f;
	hipEventElapsedTime(&gpu_time, start, stop);

	// copy data from gpu to cpu
	clock_t tohost_start = clock();
	hipMemcpy(m_r, MR, r_num * sizeof(float), hipMemcpyDeviceToHost);
	clock_t tohost_end = clock();

	cout << endl << "*******************************************" << endl
		<< "Matrix A size : " << m << " * " << n << endl
		<< "Matrix B size : " << n << " * " << k << endl
		<< "HostToDevice : " << (todevice_end - todevice_start) << " ms" << endl
		<< "GPUMultiTime : " << gpu_time << " ms" << endl
		<< "DeviceToHost : " << (tohost_end - tohost_start) << " ms" << endl
		<< "TotalTime : " << (tohost_end - todevice_start) << " ms" << endl
		<< "*******************************************"
		<< endl;
	
	// free memory
	hipFree(MA);
	hipFree(MB);
	hipFree(MR);

	return;
}

/*
* @func: matrix multiplication on gpu use shared memory
*
* @para: m_a  left multi matrix
*		 m_b  right multi matrix
*		 m_r  result matrix
*		 m    left matrix rows
*        n    left matrix cols
*        k    right matrix cols
*/
template<int BLOCK_SIZE>
__global__ void matrixMulOnGPUWithShared(float* m_a, float* m_b, float* m_r, unsigned int m, unsigned int n, unsigned int k)
{
	// thread location
	int block_x = blockIdx.x;
	int block_y = blockIdx.y;
	int thread_x = threadIdx.x;
	int thread_y = threadIdx.y;

	if ((thread_y + block_y * blockDim.y) * k + block_x * blockDim.x + thread_x >= m * k)
	{
		return;
	}

	// blockDim.x == blockDim.y == BLOCK_SIZE here
	const int begin_a = block_y * blockDim.y * n;
	const int end_a = begin_a + n - 1;
	const int step_a = blockDim.x;

	const int begin_b = block_x * blockDim.x;
	const int step_b = blockDim.y * k;

	float result_temp = 0.0f;

	for (int index_a = begin_a, int index_b = begin_b; index_a < end_a; index_a += step_a, index_b += step_b)
	{
		// shared memory
		__shared__ float SubMat_A[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float SubMat_B[BLOCK_SIZE][BLOCK_SIZE];

		// copy data to shared memory
		SubMat_A[thread_y][thread_x] = m_a[index_a + thread_y * n + thread_x];
		SubMat_B[thread_y][thread_x] = m_b[index_b + thread_y * k + thread_x];

		__syncthreads();

		for (int i = 0; i < BLOCK_SIZE; ++i)
		{
			result_temp += SubMat_A[thread_y][i] * SubMat_B[i][thread_x];
		}

		__syncthreads();
	}

	int begin_result = block_y * blockDim.y * k + begin_b;
	m_r[begin_result + thread_y * k + thread_x] = result_temp;
}

/*
* @func: matrix multiplication on gpu use shared memory
*
* @para: m_a  left multi matrix
*		 m_b  right multi matrix
*		 m_r  result matrix
*		 m    left matrix rows
*        n    left matrix cols
*        k    right matrix cols
*/
__host__ void matrixMulUseGPUWithShared(float* m_a, float* m_b, float* m_r, unsigned int m, unsigned int n, unsigned int k)
{
	if (m <= 0 || n <= 0 || k <= 0)
	{
		cout << "Matrix size error !" << endl;
		return;
	}
	float *MA, *MB, *MR;

	unsigned int a_num = m * n;
	unsigned int b_num = n * k;
	unsigned int r_num = m * k;

	// Allocated memory
	hipMalloc((void**)&MA, a_num * sizeof(float));
	hipMalloc((void**)&MB, b_num * sizeof(float));
	hipMalloc((void**)&MR, r_num * sizeof(float));

	// copy data from cpu to gpu
	clock_t todevice_start = clock();
	hipMemcpy(MA, m_a, a_num * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(MB, m_b, b_num * sizeof(float), hipMemcpyHostToDevice);
	clock_t todevice_end = clock();

	// define grids and blocks size
	unsigned int thread_num = min(r_num, THREAD_PER_BLOCK);
	unsigned int block_num = (r_num % thread_num != 0) ? (r_num / thread_num + 1) : (r_num / thread_num);

	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, NULL);
	matrixMulOnGPUWithShared<32> << <grid, block >> > (MA, MB, MR, m, n, k);
	hipDeviceSynchronize();
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	float gpu_time = 0.0f;
	hipEventElapsedTime(&gpu_time, start, stop);


	// copy data from gpu to cpu
	clock_t tohost_start = clock();
	hipMemcpy(m_r, MR, r_num * sizeof(float), hipMemcpyDeviceToHost);
	clock_t tohost_end = clock();

	cout << endl << "*******************************************" << endl
		<< "Matrix A size : " << m << " * " << n << endl
		<< "Matrix B size : " << n << " * " << k << endl
		<< "HostToDevice : " << (todevice_end - todevice_start) << " ms" << endl
		<< "GPUMultiTimeShared : " << gpu_time << " ms" << endl
		<< "DeviceToHost : " << (tohost_end - tohost_start) << " ms" << endl
		<< "TotalTime : " << (tohost_end - todevice_start) << " ms" << endl
		<< "*******************************************"
		<< endl;

	// free memory
	hipFree(MA);
	hipFree(MB);
	hipFree(MR);

	return;
}



/*
* @func: matrix multiplication on gpu use reduction algorithm
*
* @para: m_a  left multi matrix
*		 m_b  right multi matrix
*		 m_r  result matrix
*		 m    left matrix rows
*        n    left matrix cols
*        k    right matrix cols
*/
__host__ void matrixMulUseGPUWithReduction(float* m_a, float* m_b, float* m_r, unsigned int m, unsigned int n, unsigned int k)
{
	if (m <= 0 || n <= 0 || k <= 0)
	{
		cout << "Matrix size error !" << endl;
		return;
	}
	vector<vector<float>> a;
	vector<vector<float>> b;

	vector<float> a_tmp;
	vector<float> b_tmp;
	for (int i = 0; i < m; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			a_tmp.push_back(m_a[i * n + j]);
		}
		a.push_back(a_tmp);
		a_tmp.clear();
	}
	for (int i = 0; i < k; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			b_tmp.push_back(m_b[j * k + i]);
		}
		b.push_back(b_tmp);
		b_tmp.clear();
	}
	clock_t start = clock();
	thrust::device_vector<float> result(n);
	for (int i = 0; i < m; ++i)
	{
		for (int j = 0; j < k; ++j)
		{
			thrust::device_vector<float> row(a[i].begin(), a[i].end());
			thrust::device_vector<float> col(b[j].begin(), b[j].end());

			thrust::transform(row.begin(), row.end(), col.begin(), result.begin(), thrust::multiplies<float>());

			m_r[i * k + j] = thrust::reduce(result.begin(),result.end());
		}
	}
	clock_t end = clock();


	cout << endl << "*******************************************" << endl
		<< "Matrix A size : " << m << " * " << n << endl
		<< "Matrix B size : " << n << " * " << k << endl
		<< "GPUMultiTimeReduction : " << (end - start) << " ms" << endl
		<< "*******************************************"
		<< endl;

	return;
}

/*
* @func: matrix multiplication on gpu use cublas
*
* @para: m_a  left multi matrix
*		 m_b  right multi matrix
*		 m_r  result matrix
*		 m    left matrix rows
*        n    left matrix cols
*        k    right matrix cols
*/
__host__ void matrixMulUseGPUWithCublas(float* m_a, float* m_b, float* m_r, unsigned int m, unsigned int n, unsigned int k)
{
	if (m <= 0 || n <= 0 || k <= 0)
	{
		cout << "Matrix size error !" << endl;
		return;
	}
	float *MA, *MB, *MR;

	unsigned int a_num = m * n;
	unsigned int b_num = n * k;
	unsigned int r_num = m * k;

	// Allocated memory
	hipMalloc((void**)&MA, a_num * sizeof(float));
	hipMalloc((void**)&MB, b_num * sizeof(float));
	hipMalloc((void**)&MR, r_num * sizeof(float));

	// copy data from cpu to gpu
	clock_t todevice_start = clock();
	hipMemcpy(MA, m_a, a_num * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(MB, m_b, b_num * sizeof(float), hipMemcpyHostToDevice);
	clock_t todevice_end = clock();

	float alpha = 1.0f;
	float beta = 0.0f;
	hipblasHandle_t handle;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipblasCreate(&handle);
	hipEventRecord(start, NULL);
	hipblasSgemm(handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		k,
		m,
		n,
		&alpha,
		MB,
		k,
		MA,
		n,
		&beta,
		MR,
		k);
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	float cublas_time = 0.0f;
	hipEventElapsedTime(&cublas_time, start, stop);

	// copy data from gpu to cpu
	clock_t tohost_start = clock();
	hipMemcpy(m_r, MR, r_num * sizeof(float), hipMemcpyDeviceToHost);
	clock_t tohost_end = clock();

	cout << endl << "*******************************************" << endl
		<< "Matrix A size : " << m << " * " << n << endl
		<< "Matrix B size : " << n << " * " << k << endl
		<< "HostToDevice : " << (todevice_end - todevice_start) << " ms" << endl
		<< "GPUMultiTimeCublas : " << cublas_time << " ms" << endl
		<< "DeviceToHost : " << (tohost_end - tohost_start) << " ms" << endl
		<< "TotalTime : " << (tohost_end - todevice_start) << " ms" << endl
		<< "*******************************************"
		<< endl;

	// free memory
	hipFree(MA);
	hipFree(MB);
	hipFree(MR);

	return;
}

//************* MAIN FUNCTION ***************//
int main()
{
// 1�� Defining variables
	// left and right matrix size
	// m: left row  
	// n: left col|right row
	// k: right col
	unsigned int m, n, k;
	// left,right,result matrix pointer
	float *matrix_a, *matrix_b, *matrix_r;
	// mode
	int mode;

// 2��Assignment of variables
	cout << "Please select matrix multiply mode :" << endl
		<< "1��Naive CPU" << endl
		<< "2��Eigen " << endl
		<< "3��Naive GPU" << endl
		<< "4��GPU Shared Memory" << endl
		<< "5��GPU Reduction" << endl
		<< "6��GPU Cublas" << endl;

	cin >> mode;
	cout << endl << "Please input the size of left and right matrix : (m n k)" << endl;
	while (n <= 0 || m <= 0 || k <= 0)
	{
		cin >> m >> n >> k;
		if (n <= 0 || m <= 0 || k <= 0)
		{
			cout << "Matrix size must be Positive number,please input again" << endl;
		}
	}

// 3��Generate matrix
	genMatrixValue(&matrix_a, &matrix_b, &matrix_r, m, n, k);

// 4��Matrix multiply
	switch (mode)
	{
	// 4.1��matrix multiply on CPU
	case 1:
		matrixMulOnCPU(matrix_a, matrix_b, matrix_r, m, n, k);
		break;
	// 4.2��matrix multiply use Eigen
	case 2:
		matrixMulUseEigen(matrix_a, matrix_b, matrix_r, m, n, k);
		break;
	// 4.3��matrix multiply on GPU
	case 3:
		matrixMulUseGPU(matrix_a, matrix_b, matrix_r, m, n, k);
		break;
	// 4.4��matrix multiply on GPU with Shared memory
	case 4:
		matrixMulUseGPUWithShared(matrix_a, matrix_b, matrix_r, m, n, k);
		break;
	// 4.5��matrix multiply on GPU with Reduction
	case 5:
		matrixMulUseGPUWithReduction(matrix_a, matrix_b, matrix_r, m, n, k);
		break;
	// 4.6��matrix multiply on GPU with Cublas
	case 6:
		matrixMulUseGPUWithCublas(matrix_a, matrix_b, matrix_r, m, n, k);
		break;
	default:
		break;
	}
	
	return 0;
}